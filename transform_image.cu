#include "hip/hip_runtime.h"
/*
MIT License

Copyright (c) 2022 Alan Lira

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

#include <ctype.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb/stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb/stb_image_write.h"

void string_to_lower_case(char *string) {
    for (int i = 0; string[i]; i++) {
        string[i] = tolower(string[i]);
    }
}

void string_to_upper_case(char *string) {
    for (int i = 0; string[i]; i++) {
        string[i] = toupper(string[i]);
    }
}

void nearest_neighbor_interpolation(unsigned char *input_image,
                                    unsigned char *output_image,
                                    int input_image_width,
                                    int output_image_width,
                                    int output_image_height,
                                    float scale_x,
                                    float scale_y,
                                    int image_channels) {
    for (int y = 0; y < output_image_height; y++) {
        for (int x = 0; x < output_image_width; x++) {
            int x_nearest = (int) x / scale_x;
            int y_nearest = (int) y / scale_y;
            for (int c = 0; c < image_channels; c++) {
                int index_input = (input_image_width * y_nearest * image_channels) + (x_nearest * image_channels);
                int index_output = (output_image_width * y * image_channels) + (x * image_channels);
                output_image[index_output + c] = input_image[index_input + c];
            }
        }
    }
}

void execute_on_cpu(unsigned char *input_image,
                    unsigned char *output_image,
                    int input_image_width,
                    int output_image_width,
                    int output_image_height,
                    float scale_x,
                    float scale_y,
                    int image_channels) {
    // Execute the Nearest Neighbor Interpolation (Image Scaling) on the Host (CPU).
    nearest_neighbor_interpolation(input_image,
                                   output_image,
                                   input_image_width,
                                   output_image_width,
                                   output_image_height,
                                   scale_x,
                                   scale_y,
                                   image_channels);
}

// Wrapper for CUDA Functions Calls.
#define CUDA_CHECK(call) \
    if ((call) != hipSuccess) { \
        hipError_t cuda_error = hipGetLastError(); \
	printf("The Following CUDA Error Occurred: %s.\n", hipGetErrorString(cuda_error)); \
        exit(4); \
    }

__global__ void nearest_neighbor_interpolation_kernel(unsigned char *input_image,
                                                      unsigned char *output_image,
                                                      int input_image_width,
                                                      int output_image_width,
                                                      int output_image_height,
                                                      float scale_x,
                                                      float scale_y,
                                                      int image_channels) {
    int y = (blockDim.y * blockIdx.y) + threadIdx.y;
    int x = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (y < output_image_height && x < output_image_width) {
        int x_nearest = (int) x / scale_x;
        int y_nearest = (int) y / scale_y;
        for (int c = 0; c < image_channels; c++) {
            int index_input = (input_image_width * y_nearest * image_channels) + (x_nearest * image_channels);
            int index_output = (output_image_width * y * image_channels) + (x * image_channels);
            output_image[index_output + c] = input_image[index_input + c];
        }
    }
}

void execute_on_gpu_with_cuda(unsigned char *input_image_host,
                              unsigned char *output_image_host,
                              int input_image_width,
                              int input_image_height,
                              int input_image_channels,
                              int output_image_width,
                              int output_image_height,
                              int output_image_channels,
                              float scale_x,
                              float scale_y) {
    // Input Image's Data Memory Alloc for the Device (GPU).
    unsigned char *input_image_device;
    CUDA_CHECK(hipMalloc(&input_image_device,
                          sizeof(unsigned char) * input_image_width * input_image_height * input_image_channels));
    // Copy the Input Image's Data From Host (CPU) to Device (GPU).
    CUDA_CHECK(hipMemcpy(input_image_device,
                          input_image_host,
                          sizeof(unsigned char) * input_image_width * input_image_height * input_image_channels,
                          hipMemcpyHostToDevice));
    // Output Image's Data Memory Alloc for the Device (GPU).
    unsigned char *output_image_device;
    CUDA_CHECK(hipMalloc(&output_image_device,
                          sizeof(unsigned char) * output_image_width * output_image_height * output_image_channels));
    // Set the Number of Threads per Block (dimBlock) and the Number of Blocks (dimGrid).
    dim3 dimBlock(32, 32);
    dim3 dimGrid(ceil((float) output_image_width / dimBlock.x), ceil((float) output_image_height / dimBlock.y));
    // Execute the Nearest Neighbor Interpolation Kernel (Image Scaling) on the Device (GPU).
    nearest_neighbor_interpolation_kernel<<<dimGrid, dimBlock>>>(input_image_device,
                                                                 output_image_device,
                                                                 input_image_width,
                                                                 output_image_width,
                                                                 output_image_height,
                                                                 scale_x,
                                                                 scale_y,
                                                                 input_image_channels);
    // Wait for the GPU to Finish the Kernel Execution.
    hipDeviceSynchronize();
    // Copy the Output Image's Data From Device (GPU) to Host (CPU).
    CUDA_CHECK(hipMemcpy(output_image_host,
                          output_image_device,
                          sizeof(unsigned char) * output_image_width * output_image_height * output_image_channels,
                          hipMemcpyDeviceToHost));
    // Free the Memory Allocated for the Device (GPU).
    CUDA_CHECK(hipFree(input_image_device));
    CUDA_CHECK(hipFree(output_image_device));
}

int main(int argc, char **argv) {
    // Begin.
    // Check the Number of Arguments Provided (Expected: argc = 7).
    if (argc != 7) {
        printf("USAGE: transform_image <CPU | GPU_CUDA> <Input_Image_Path> <Output_Image_Width> <Output_Image_Height> <Output_Image_Channels> <Output_Image_Path>\n");
	exit(1);
    }
    // Parse the Arguments Provided (argv).
    char *execution_type = argv[1];
    char *input_image_file = argv[2];
    int output_image_width = atoi(argv[3]);
    int output_image_height = atoi(argv[4]);
    int output_image_channels = atoi(argv[5]);
    char *output_image_file = argv[6];
    // Initialize the Input Image's Variables (Width, Height, and Channels).
    // Channels = 1 --> Grey
    // Channels = 2 --> Grey, Alpha
    // Channels = 3 --> Red, Green, Blue
    // Channels = 4 --> Red, Green, Blue, Alpha
    int input_image_width = 0, input_image_height = 0, input_image_channels = 0;
    // Load the Input Image Using stb_image and Return the Resulting Data and Its Properties:
    // Width, Height, and Number of 8-Bit Components (Channels) per Pixel in the Image.
    int input_image_desired_channels = 0; // Default Value: 0 (All).
    unsigned char *input_image_host = stbi_load(input_image_file,
                                                &input_image_width,
                                                &input_image_height,
                                                &input_image_channels,
                                                input_image_desired_channels);
    // Check if the Input Image Was Succesfully Loaded on the Host (CPU).
    if (input_image_host == NULL) {
        printf("ERROR When Trying to Load the Input Image '%s'!\n", input_image_file);
        exit(2);
    }
    // Check if the Number of Channels on Input and Output Images Are Equal.
    if (input_image_channels != output_image_channels) {
        printf("The Number of Channels on the Input and Output Images Must be Equal! (Input Image's Channels: %d)\n", input_image_channels);
        exit(3);
    }
    // Input Image's Summary.
    printf("Input Image Loaded: '%s' --> Width = %dpx, Height = %dpx, and Channels = %d.\n",
           input_image_file,
           input_image_width,
           input_image_height,
           input_image_channels);
    // Output Image's Data Memory Alloc for the Host (CPU).
    unsigned char *output_image_host;
    output_image_host = (unsigned char *) malloc(sizeof(unsigned char) * output_image_width * output_image_height * output_image_channels);
    // Calculate the Scaling Factors (scale_x and scale_y).
    // A Scale Factor < 1 Indicates Image Shrinking;
    // A Scale Factor > 1 Indicates Image Stretching.
    float scale_x = (float) output_image_width / input_image_width;
    float scale_y = (float) output_image_height / input_image_height;
    // Set the Processing Time Variables.
    clock_t t = clock();
    double runtime_in_seconds = 0.0;
    // Get the Execution Type String and Transform It (Lower Case).
    string_to_lower_case(execution_type);
    // Process the Input Image (Using CPU Only or Using GPU With CUDA).
    if (strcmp(execution_type, "cpu") == 0) {
        printf("Processing the Input Image '%s' Using CPU Only (Scaling Factor: scale_x = %.2f, scale_y = %.2f)...\n",
               input_image_file,
               scale_x,
               scale_y);
        execute_on_cpu(input_image_host,
                       output_image_host,
                       input_image_width,
                       output_image_width,
                       output_image_height,
                       scale_x,
                       scale_y,
                       input_image_channels);
    } else if (strcmp(execution_type, "gpu_cuda") == 0) {
        printf("Processing the Input Image '%s' Using GPU With CUDA (Scaling Factor: scale_x = %.2f, scale_y = %.2f)...\n",
               input_image_file,
               scale_x,
               scale_y);
        execute_on_gpu_with_cuda(input_image_host,
                                 output_image_host,
                                 input_image_width,
                                 input_image_height,
                                 input_image_channels,
                                 output_image_width,
                                 output_image_height,
                                 output_image_channels,
                                 scale_x,
                                 scale_y);
    }
    // Calculate the Input Image Processing Time.
    t = clock() - t;
    runtime_in_seconds = (double) t / CLOCKS_PER_SEC;
    // Get the Execution Type String and Transform It (Upper Case).
    string_to_upper_case(execution_type);
    // Print the Input Image Processing Time.
    printf("Input Image Processing Time Using %s: %.2f ms (%.2f s)\n",
           execution_type,
           (runtime_in_seconds * 1000),
           runtime_in_seconds);
    // Write the Output Image JPG File Using stb_image.
    int output_image_quality = 100;
    stbi_write_jpg(output_image_file,
                   output_image_width,
                   output_image_height,
                   output_image_channels,
                   output_image_host,
                   output_image_quality);
    // Output Image's Summary.
    printf("Output Image Saved: '%s' --> Width = %dpx, Height = %dpx, Channels = %d, and Quality = %d%%.\n",
           output_image_file,
           output_image_width,
           output_image_height,
           output_image_channels,
           output_image_quality);
    // Free the Memory Allocated for the Host (CPU).
    stbi_image_free(input_image_host);
    free(output_image_host);
    // End.
    exit(0);
}

